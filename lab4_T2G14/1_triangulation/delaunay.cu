#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>
#include "hip/hip_runtime.h"

#define pixel(i, j, w)  (((j)*(w)) +(i))
#define THREADSPERBLOCK 1024

int max_num_triangles;
#define B 16

/* A point in 2D space */
struct Point {
    double x;
    double y;
    double value;
};

/* A triangle defined by three points */
struct Triangle {
    struct Point p1;
    struct Point p2;
    struct Point p3;
};

/* Helper function to output the triangles in the Delaunay Triangulation */
void print_triangles(struct Triangle * triangles, int num_triangles) {
    for (int i = 0; i < num_triangles; i++) {
        printf("(%lf, %lf) (%lf, %lf) (%lf, %lf)\n", 
            triangles[i].p1.x, triangles[i].p1.y,
            triangles[i].p2.x, triangles[i].p2.y,
            triangles[i].p3.x, triangles[i].p3.y);   
    }
}

/* Helper function to calculate the distance between two points */
double distance(struct Point * p1, struct Point * p2) {
    double dx = (*p1).x - (*p2).x;
    double dy = (*p1).y - (*p2).y;
    return sqrt(dx*dx + dy*dy);
}

/* Helper function to check if a triangle is clockwise */
int is_ccw(struct Triangle * t) {
    double ax = (*t).p2.x - (*t).p1.x;
    double ay = (*t).p2.y - (*t).p1.y;
    double bx = (*t).p3.x - (*t).p1.x;
    double by = (*t).p3.y - (*t).p1.y;

    double area = ax * by - ay * bx;
    return area > 0;
}

/* Helper function to check if a point is inside a circle defined by three points */
int inside_circle(struct Point * p, struct Triangle * t) {
//      | ax-dx, ay-dy, (ax-dx)² + (ay-dy)² |
//det = | bx-dx, by-dy, (bx-dx)² + (by-dy)² |
//      | cx-dx, cy-dy, (cx-dx)² + (cy-dy)² |

    int clockwise = is_ccw(t);
    
    double ax = (*t).p1.x - (*p).x;
    double ay = (*t).p1.y - (*p).y;
    double bx = (*t).p2.x - (*p).x;
    double by = (*t).p2.y - (*p).y;
    double cx = (*t).p3.x - (*p).x;
    double cy = (*t).p3.y - (*p).y;

    double det = ax*by + bx*cy + cx*ay - ay*bx - by*cx - cy*ax;
    det = (ax*ax + ay*ay) * (bx*cy-cx*by) -
            (bx*bx + by*by) * (ax*cy-cx*ay) +
            (cx*cx + cy*cy) * (ax*by-bx*ay);
    
    if(clockwise)
        return det > 0;
    return det<0;
}

//* Helper function to compute barycentric coordintaes of a point respect a triangle */
void barycentric_coordinates(struct Triangle * t, struct Point * p, double * alpha, double * beta, double * gamma) {
    // Compute the barycentric coordinates of the point with respect to the triangle
    (*alpha) = (((*t).p2.y - (*t).p3.y) * ((*p).x - (*t).p3.x) + ((*t).p3.x - (*t).p2.x) * ((*p).y - (*t).p3.y)) /
                  (((*t).p2.y - (*t).p3.y) * ((*t).p1.x - (*t).p3.x) + ((*t).p3.x - (*t).p2.x) * ((*t).p1.y - (*t).p3.y));
    (*beta) = (((*t).p3.y - (*t).p1.y) * ((*p).x - (*t).p3.x) + ((*t).p1.x - (*t).p3.x) * ((*p).y - (*t).p3.y)) /
                 (((*t).p2.y - (*t).p3.y) * ((*t).p1.x - (*t).p3.x) + ((*t).p3.x - (*t).p2.x) * ((*t).p1.y - (*t).p3.y));
    (*alpha) =(*alpha) > 0 ? (*alpha) : 0;
    (*alpha) =(*alpha) < 1 ? (*alpha) : 1;
    (*beta) = (*beta) > 0 ? (*beta) : 0;
    (*beta) = (*beta) < 1 ? (*beta) : 1;
    (*gamma) = 1.0 - (*alpha) - (*beta);
    (*gamma) = (*gamma) > 0 ? (*gamma) : 0;
    (*gamma) = (*gamma) < 1 ? (*gamma) : 1;
}


/* Helper function to check if a point is inside a triangle (IT CAN BE REMOVED)*/
int inside_triangle(struct Triangle * t, struct Point * p) {
    double alpha, beta, gamma;
    barycentric_coordinates(t, p, &alpha, &beta, &gamma); 
    // Check if the barycentric coordinates are positive and add up to 1
    if (alpha > 0 && beta > 0 && gamma > 0) {
        return 1;
    } else {
        return 0;
    }
}

/* Helper function to save an image */   
void save_image(char const * filename, int width, int height, double *image){

   FILE *fp=NULL;
   fp = fopen(filename,"w");
   for(int j=0; j<height; ++j){
      for(int i=0; i<width; ++i){
         fprintf(fp,"%f ", image[pixel(i,j,width)]);      
      }
      fprintf(fp,"\n");
   }
   fclose(fp);

}

/* helper function to initialize the points */
void init_points(struct Point* points, int num_points, int width, int height) {
    for(int i = 0; i < num_points; i++) {
        points[i].x =  ((double) rand() / RAND_MAX)*width;
        points[i].y =  ((double) rand() / RAND_MAX)*height;
        points[i].value = 0;//(rand() % 10000) / 100.;
        //printf("Point %d [%f,%f]=%f\n", i, points[i].x, points[i].y, points[i].value);
    }
}


/////////////////////////////////////////////
///
///         CUDA part
///
/////////////////////////////////////////////

__device__ double distance_CUDA(struct Point * p1, struct Point * p2) {
    double dx = (*p1).x - (*p2).x;
    double dy = (*p1).y - (*p2).y;
    return sqrt(dx*dx + dy*dy);
}


/*Kernel function: to be executed on the device and launched from the host*/
__global__ void count_close_points_CUDA(struct Point* points, int num_points) {
    
    int id = threadIdx.x + blockIdx.x * blockDim.x; // get gloval iter

    if(id <= num_points * num_points) return; 

    int i = (id / num_points)%num_points; 
    int j = id%num_points; 

    //printf("Indexes: i:%d and j:%d\n", threadIdx.x, threadIdx.y);
    printf("Indexes: i:%d and j:%d, total iter: %d = %d\n", i, j, id, i * num_points + j);

    if( !(i < j) ) return; 

    double dis = distance_CUDA(&points[i], &points[j]);
    if(dis <= 100){
        points[i].value++;                          //stores +1 in value
        points[j].value++; 
    }
    


}

/*Wraper function to launch the CUDA kernel to count the close points*/
void count_close_points_gpu(struct Point* points, int num_points) {
    int dim_grid = 0;                       //num_blocks
    int dim_block = 0;                      //threads/block

    //create pointer into the gpu
    struct Point* d_points;

    //allocate memory in the gpu
    hipMalloc(&d_points, sizeof(struct Point) * num_points);

    //copy memory into the gpu
    hipMemcpy(d_points, points, sizeof(struct Point) * num_points, hipMemcpyHostToDevice);          //we transfer it from CPU -> GPU

    printf("Num points: %d \n", num_points);
    
    //en una dim
    /*if(num_points <= THREADSPERBLOCK){
        dim_grid = 1;
        dim_block = num_points;
    }else{
        dim_grid = ceil((double)(num_points/1024));             // 32 x 32 = 1024
        dim_block = THREADSPERBLOCK;
    }*/
    /*
    //en 2 dimensiones
    dim_grid = ceil((double)(num_points/THREADSPERBLOCK));             // 32 x 32 = 1024

    dim3 dimGrid((num_points+31)/3);        //?????????????
    dim3 dimBlock(32, 32);

    printf("DimGrid: %d DimBlock: %d \n", dim_grid, dim_block);

    count_close_points_CUDA<<<dimGrid, dimBlock>>>(d_points, num_points);                     //(dimGrid, dimBlock) we want to iterate over every pair
    */

    int n_blocks = (int)ceil(((double)num_points * num_points)/THREADSPERBLOCK); 
    count_close_points_CUDA<<<n_blocks, THREADSPERBLOCK>>>(d_points, num_points);                     //(dimGrid, dimBlock) we want to iterate over every pair


    hipMemcpy(points, d_points, sizeof(struct Point) * num_points, hipMemcpyDeviceToHost);          //we transfer it from GPU -> CPU

    //deallocate
    hipFree(d_points);

}


__global__ void delaunay_triangulation_CUDA(struct Point* points, int num_points, struct Triangle* triangles, int* num_triangles) {

    int nt = 0;                                                 // <- creo que no sirve de nada

    int n_totalIter = threadIdx.x + blockIdx.x * blockDim.x;    //numero de la iteracion global ("entre los 3 fors")

    if(num_points * num_points * num_points < n_totalIter) return; 

    int i = n_totalIter / (num_points * num_points);            //recupera i, j y k
    int j = (n_totalIter / num_points) % num_points;
    int k = n_totalIter % num_points; 

    if( !( i < j && j < k ) ) return; //if the conditions are NOT met, end thread
    //calculate triangle
    struct Triangle triangle_new;

    triangle_new.p1 = points[i];
    triangle_new.p2 = points[j];
    triangle_new.p3 = points[k];
    
    int inside = 0;
    for(int p = 0; p < num_points; p++) {        
        inside = inside_circle(&points[p], &triangle_new);      // result is 0 or 1 --> need to adapt it to use CUDA
        if(inside) break;          
    }
    //#pragma acc wait                                          //waits all previously queued work

    if(inside == 0) {                                           //if no other point is inside the triangle

        triangles[*num_triangles] = triangle_new;               //nt is updated after the assignation
        //aqui podria haver race condition
        atomicAdd_system(num_triangles, 1);                     //atomic add +1

    } 


}

/*Wraper function to launch the CUDA kernel to compute delaunay triangulation*/
void delaunay_triangulation_gpu(struct Point* points, int num_points, struct Triangle* triangles, int* num_triangles) {

    /*
    int dim_grid, dim_block;
    dim_grid = 0;
    dim_block = 0;
    */
    struct Point* d_points;                                                                   //ptr GPU
    hipMalloc(&d_points, sizeof(struct Point) * num_points);                                 //allocate space
    hipMemcpy(d_points, points, sizeof(struct Point) * num_points, hipMemcpyHostToDevice);  //data transfer CPU -> GPU

    //repeat for triangles
    struct Triangle* d_triangles;                                                             //ptr GPU
    hipMalloc(&d_triangles, sizeof(struct Triangle) * num_points * 30);                       //allocate space//in prevois lab, max triangles were num_poits * 30 (or so)
    //no need to copy memory, since the array will be filled there

    int totalIters = num_points * num_points * num_points;                                      // num_points**3

    int* d_nt;                                                                                  //device num triangles
    int h_nt = -3550000;                                                                        // host num triangles //pongo este numero para detectar posibles errores
    hipMallocManaged(&d_nt, sizeof(int));                                                      //allocate int //lo hago con el managed porque en el tuto lo hacia así. 
    *d_nt = 0;                                                                                  //no sé si esto funciona pero estava en el tuto

    /*
    if(totalIters <= 1024){
        dim_grid = 1;
        dim_block = totalIters;
    }else{
        dim_grid = ceil((double)(totalIters/1024));
        dim_block = 1024;
    }*/

    /*
    dim3 dimGrid(dim_grid);
    dim3 dimBlock(dim_block);
    */
    int n_blocks = (int)ceil(((double)num_points * num_points * num_points)/THREADSPERBLOCK); 

    delaunay_triangulation_CUDA<<<n_blocks, THREADSPERBLOCK>>>(d_points, num_points, d_triangles, d_nt);      //entiendo que falta el block_size(?)
    //Syncronization (?)

    hipMemcpy(h_nt, d_nt, sizeof(int), hipMemcpyDeviceToHost);                               ////data transfer GPU -> CPU

    //no need to retrive points since they are not affected
    hipMemcpy(triangles, d_triangles, sizeof(struct Triangle) * h_nt, hipMemcpyDeviceToHost); //retrive only the necessary triangles

    *num_triangles = h_nt; //save the value of 
    hipFree(d_points);
    hipFree(d_triangles);


    //Display some info and print some of the triangles //delete later
    if(0 <= h_nt){
        int rnd = 5; //bad pseudoransom number generator
        printf("delaunay_triangulation_gpu finalized. Created %d triangles (neg number = error) \n", *num_triangles)
        for(rnd = 5; rnd < *num_triangles; rnd += 20 + (rnd * 73)%29) { //print some of the triangles
            printf("Triangle %d : [(%lf, %lf) (%lf, %lf) (%lf, %lf)] \n", rnd
            triangles[rnd].p1.x, triangles[rnd].p1.y,
            triangles[rnd].p2.x, triangles[rnd].p2.y,
            triangles[rnd].p3.x, triangles[rnd].p3.y);   
        }
    } else printf("There has been an error with d_nt or h_nt variables. h_nt = %d \n", h_nt); 


}


// __global__ void save_triangulation_points_CUDA(struct Point* points, int num_points, struct Triangle* triangles, int* num_triangles, double* image, int width, int height) {
    
//     int i = threadIdx.x + blockIdx.x * blockDim.x; //get position of pixel
//     int j = threadIdx.y + blockIdx.y * blockDim.y; 

//     //declare vars
//     int inside = 0;
//     struct Point pixel, *point;
//     struct Triangle* tr = NULL; 
//     double alpha, beta, gamma;

//     pixel.x = (double)i; 
//     pixel.y = (double)j; 
//     pixel.value = 0;

//     image[pixel(i, j, width)] = -1; //set deafult value


//     for(int k = 0; k < num_triangles; k++){             //recorre todos los triangulos
//         tr = &triangles[k]; 
//         barycentric_coordinates(tr, &pixel, &alpha, &beta, &gamma); 
//         if(0 < alpha && 0 < beta && 0 < gamma){ //if inside triangle
//             image[pixel(i, j, width)] = tr->p1.value * alpha + tr->p2.value * beta + tr->p3.value * gamma;   //sets new value
//             break; // podria ser un return
//         }
//     }



// }

// __global__ void save_BlackBox_CUDA(struct Point* points, int num_points, double* image, int width, int height) {

//     int k = threadIdx.x + blockIdx.x * blockDim.x; 

//     int _x = points[k].x; //get coord of point
//     int _y = points[k].y; 
    
//     int radius = 2; // Total size = (2 * radius + 1)^2

//     //square of size 5

//     for(int i = _x - radius;  i <= _x + radius; i++) { //in a box
//         for(int j = _y - radius; j <= _y + radius; j++) {
//             if(0 <= i && 0 <= j && i < width && j < height) { //if possible
//                 image[(pixel(i, j, width))] = 101.0; //draw black pixel
//             }
//         }
//     }

// }



/*Wraper function to launch the CUDA kernel to compute delaunay triangulation. 
Remember to store an image of int's between 0 and 100, where points store 101, and empty areas -1, and points inside triangle the average of value */

/*
void save_triangulation_image_gpu(struct Point* points, int num_points, struct Triangle* triangles, int num_triangles, int width, int height) {
    
    //create structures
    int size = width * height;
    double* image = (double*) malloc(sizeof(double)*size);

    //copy points to gpu
    struct Point* cudaPoints; //ptr GPU
    hipMalloc(&cudaPoints, sizeof(struct Point) * num_points); //allocate space
    hipMemcpy(cudaPoints, points, sizeof(struct Point) * num_points, hipMemcpyHostToDevice); //data transfer

    //copy triangles to gpu
    struct Triangle* cudaTriangles; //ptr GPU
    hipMalloc(&cudaTriangles, sizeof(struct Triangle) * num_triangles); //allocate space
    hipMemcpy(cudaTriangles, triangles, sizeof(struct Triangle) * num_triangles, hipMemcpyHostToDevice); //data transfer

    double* cudaImage; //ptr GPU
    hipMalloc(&cudaImage, sizeof(double) * size); //allocate space
    //data created in gpu

    //usamos un thread en la gpu por pixel
    save_triangulation_points_CUDA<<<width, height>>>(cudaPoints, num_points, cudaTriangles, num_triangles, cudaImage, width, height);                                

    hipFree(cudaTriangles); //not needed anymore

    //wait for next kernel
    //keep image in gpu, no need to move it
    //also keep points there

    save_BlackBox_CUDA<<<num_points>>>(cudaPoints, num_points, cudaImage, width, height); 

    hipMemcpy(image, cudaImage, sizeof(double) * size, hipMemcpyDeviceToHost); //retrive image

    hipFree(cudaPoints); 
    hipFree(cudaImage); 


    //write image
    save_image("image.txt", width, height, image);

    //free structures
    free(image);
    
}*/

void printCudaInfo() {
    int devNo = 0;
    printf("\n------------------------------------------------------------------\n");
    hipDeviceProp_t iProp;
    hipGetDeviceProperties(&iProp, devNo);
    printf("Maximum grid size is: (");
    for (int i = 0; i < 3; i++)
        printf("%d, ", iProp.maxGridSize[i]);
    printf(")\n");
    printf("Maximum block dim is: (");
    for (int i = 0; i < 3; i++)
        printf("%d, ", iProp.maxThreadsDim[i]);
    printf(")\n");
    printf("Max threads per block: %d\n", iProp.maxThreadsPerBlock);
    printf("------------------------------------------------------------------\n\n");
}

extern "C" int delaunay(int num_points, int width, int height) {
    printCudaInfo();
    
    double start, end;

    max_num_triangles = num_points*30;
    struct Point * points = (struct Point *) malloc(sizeof(struct Point)*num_points);
    struct Triangle * triangles = (struct Triangle *) malloc(sizeof(struct Triangle)*max_num_triangles);
    printf("Maximum allowed number of triangles = %d\n", num_points*30);
    
    init_points(points, num_points, width, height);

    //start = omp_get_wtime();                            //we need to use cudaEvent
    printf("Hi\n");
    count_close_points_gpu(points, num_points);
    //hipDeviceSynchronize();
    //end = omp_get_wtime();
    
    printf("Counting close points: %f\n", end-start);

    printf("Some points -> point[5]: %f\n", points[5].value);

    int num_triangles = 0;
    //start = omp_get_wtime();
    //delaunay_triangulation_gpu(points, num_points, triangles, &num_triangles);
    //end = omp_get_wtime();
    printf("Delaunay triangulation: %f\n", end-start);

    printf("Number of generated triangles = %d\n", num_triangles);
    //print_triangles(triangles, num_triangles);

    //start = omp_get_wtime();
    //hipEventRecord(start);
    //save_triangulation_image_gpu(points, num_points, triangles, num_triangles, width, height);
    //end = omp_get_wtime();
    printf("Generate image: %f\n", end-start);

    //Free memory
    free(points);
    free(triangles);

    return 0;
}